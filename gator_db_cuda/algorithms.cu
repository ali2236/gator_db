#include "hip/hip_runtime.h"
#include "algorithms.cuh"
#include "hip/hip_runtime.h"
#include "hip/device_functions.h"
#include <stdio.h>

__device__ bool compare_rows(int* a, int* b, int width, int* compare_fields, int compare_count, int asc) {
	int acc = 0;
	for (size_t i = 0; i < compare_count; i++)
	{
		int c = compare_fields[i];
		acc += a[c] - b[c];
	}
	if (asc) {
		return acc < 0;
	}
	else {
		return acc > 0;
	}
};

__global__ void oddEvenSort(int* table, int width, int length, int* sort_keys, int sort_keys_len, int asc) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int phase, i;
	int temp;

	for (phase = 0; phase < length; phase++) {
		if (phase % 2 == 0) {
			if (tid % 2 == 0) {
				if (tid < length - 1 && compare_rows(table + (tid * width), table + ((tid + 1) * width), width, sort_keys, sort_keys_len, asc)) {
					// swap
					for (size_t j = 0; j < width; j++)
					{
						temp = table[tid * width + j];
						table[tid * width + j] = table[(tid + 1) * width + j];
						table[(tid + 1) * width + j] = temp;
					}
				}
			}
		}
		else {
			if (tid % 2 != 0) {
				if (tid < length - 1 && compare_rows(table + (tid * width), table + ((tid + 1) * width), width, sort_keys, sort_keys_len, asc)) {
					// swap
					for (size_t j = 0; j < width; j++)
					{
						temp = table[tid * width + j];
						table[tid * width + j] = table[(tid + 1) * width + j];
						table[(tid + 1) * width + j] = temp;
					}
				}
			}
		}
		__syncthreads();
	}
}

hipError_t sort(int* data, int width, int length, int* order_cols, int order_cols_count, int asc) {
	int* device_table;
	int* device_sort_keys;
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) return cudaStatus;

	cudaStatus = hipMalloc((void**)&device_table, sizeof(int) * length * width);
	if (cudaStatus != hipSuccess) return cudaStatus;

	cudaStatus = hipMalloc((void**)&device_sort_keys, sizeof(int) * order_cols_count);
	if (cudaStatus != hipSuccess) return cudaStatus;

	cudaStatus = hipMemcpy(device_table, data, sizeof(int) * length * width, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) return cudaStatus;

	cudaStatus = hipMemcpy(device_sort_keys, order_cols, sizeof(int) * order_cols_count, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) return cudaStatus;

	oddEvenSort<<<1,1>>>(device_table, width, length, device_sort_keys, order_cols_count, asc);
	if (cudaStatus != hipSuccess) return cudaStatus;

	cudaStatus = hipMemcpy(data, device_table, sizeof(int) * length * width, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) return cudaStatus;

	cudaStatus = hipFree(&device_table);
	if (cudaStatus != hipSuccess) return cudaStatus;

	cudaStatus = hipFree(&device_sort_keys);
	if (cudaStatus != hipSuccess) return cudaStatus;
}

void limit(int* table, int& length, int new_length) {
	length = new_length > length ? length : new_length;
}